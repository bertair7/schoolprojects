#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include <numeric>
#include <stdlib.h>

#define BLUR_SIZE 5
#define USE_2D 0

//define the storage for the blur kernel in GPU Constant Memory
__constant__ float M_d[BLUR_SIZE];

cv::Mat imageRGBA;
cv::Mat imageGrey;
cv::Mat image;
uchar4 *d_rgbaImage__;
uchar4 *d_greyImage__;

size_t numRows() { return imageRGBA.rows; }
size_t numCols() { return imageRGBA.cols; }
const long numPixels = numRows() * numCols();

//returns a pointer to an RGBA version of the input image
//and a pointer to the single channel grey-scale output
//on both the host and device
void preProcess(uchar4 **inputImage, uchar4 **greyImage,
				uchar4 **d_rgbaImage, uchar4 **d_greyImage,
				const std::string &filename) {
	//make sure the context initializes ok
	hipFree(0);
	//Read Image into an OpenCV Matrix
	image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
	if (image.empty()) {
		std::cerr << "Couldn't open file: " << filename << std::endl;
		exit(1);
	}
	cv::cvtColor(image, imageRGBA, CV_BGR2RGBA);

	//allocate memory for the output
	imageRGBA.copyTo(imageGrey);
	//This shouldn't ever happen given the way the images are created
	//at least based upon my limited understanding of OpenCV, but better to check
	if (!imageRGBA.isContinuous() || !imageGrey.isContinuous()) {
		std::cerr << "Images aren't continuous!! Exiting." << std::endl;
		exit(1);
	}
	*inputImage = (uchar4 *)imageRGBA.ptr<unsigned char>(0);
	*greyImage  = (uchar4 *)imageGrey.ptr<unsigned char>(0);
	const size_t numPixels = numRows() * numCols();

	//allocate memory on the device for both input and output
   hipMalloc((void**) d_rgbaImage, numPixels*sizeof(uchar4));
   hipMalloc((void**) d_greyImage, numPixels*sizeof(uchar4));
   
   //make sure no memory is left laying around
   hipMemset(*d_greyImage, 0, numPixels*sizeof(uchar4));

	//copy input array to the GPU
	hipMemcpy(*d_rgbaImage, *inputImage, numPixels*sizeof(uchar4), hipMemcpyHostToDevice);

	d_rgbaImage__ = *d_rgbaImage;
	d_greyImage__ = *d_greyImage;
}

void postProcess(const std::string& output_file) {
	//copy the output back to the host
   const int num_pixels = numRows() * numCols();
	hipMemcpy(imageGrey.ptr<unsigned char>(0), d_greyImage__, num_pixels*sizeof(uchar4), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	//change in color space required by OpenCV	
	cv::cvtColor(imageGrey, imageGrey, CV_BGR2RGBA);
	//output the image to a file
	cv::imwrite(output_file.c_str(), imageGrey);
	//display the output image (will only work if you are on the lab machines)
	cv::imshow("Output Image", imageGrey);
	cv::waitKey(0);
	////cleanup
	hipFree(d_rgbaImage__);
	hipFree(d_greyImage__);

}

__global__
void conv1DRow(uchar4* const rgbaImage, uchar4* const greyImage, int numRows, int numCols)
{
	// Original Image is an array, each element of the array has 4 components .z -> R (red); .y -> G (Green) ; .x -> B (blue); .w -> A (alpha, you can ignore this one)
	//so you can read one input pixel like this:
   //B = rgbaImage[currow * numCols + curcol].x*M_d[curcolkernel]; 
	//G = rgbaImage[currow * numCols + curcol].y*M_d[curcolkernel];
	//R = rgbaImage[currow * numCols + curcol].z*M_d[curcolkernel];
	int indX = (blockIdx.x * blockDim.x) + threadIdx.x;
   int indY = (blockIdx.y * blockDim.y) + threadIdx.y;
   int curX;

   float blurX = 0.0;
   float blurY = 0.0;
   float blurZ = 0.0;
   float blurW = 1.0;

   if (indX >= 0 && indX < numCols && indY >= 0 && indY < numRows) {
      for (int i = -2; i <= 2; i++) {
         curX = indX + i;
      
         if (curX >= 0 && curX < numCols) {
            blurX += rgbaImage[indY * numCols + curX].x * M_d[i+2];
            blurY += rgbaImage[indY * numCols + curX].y * M_d[i+2];
            blurZ += rgbaImage[indY * numCols + curX].z * M_d[i+2];
         }
      }
      greyImage[indY * numCols + indX].x = (int)blurX;
      greyImage[indY * numCols + indX].y = (int)blurY;
      greyImage[indY * numCols + indX].z = (int)blurZ;
      greyImage[indY * numCols + indX].w = (int)blurW;
   }
}

__global__
void conv1DCol(uchar4* const rgbaImage, uchar4* const greyImage, int numRows, int numCols)
{
	int indX = (blockIdx.x * blockDim.x) + threadIdx.x;
   int indY = (blockIdx.y * blockDim.y) + threadIdx.y;
   int curY;

   float blurX = 0.0;
   float blurY = 0.0;
   float blurZ = 0.0;
   float blurW = 1.0;

   if (indX >= 0 && indX < numCols && indY >= 0 && indY < numRows) {
      for (int i = -2; i <= 2; i++) {
         curY = indY + i;
      
         if (curY >= 0 && curY < numRows) {
            blurX += rgbaImage[curY * numCols + indX].x * M_d[i+2];
            blurY += rgbaImage[curY * numCols + indX].y * M_d[i+2];
            blurZ += rgbaImage[curY * numCols + indX].z * M_d[i+2];
         }
      }
      greyImage[indY * numCols + indX].x = (int)blurX;
      greyImage[indY * numCols + indX].y = (int)blurY;
      greyImage[indY * numCols + indX].z = (int)blurZ;
      greyImage[indY * numCols + indX].w = (int)blurW;
   }
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage,
							uchar4 * d_rgbaImage,
							uchar4*  d_greyImage,
							size_t numRows,
							size_t numCols)
{
	float M_h[BLUR_SIZE]={0.0625,0.25,0.375,0.25,0.0625}; 
	hipMemcpyToSymbol(HIP_SYMBOL(M_d), M_h, BLUR_SIZE*sizeof(float)); //allocates/copy to Constant Memory on the GPU
	//temp image
	uchar4 *d_greyImageTemp;
	hipMalloc((void **)&d_greyImageTemp, sizeof(uchar4) * numRows*numCols);
	hipMemset(d_greyImageTemp, 0, numRows*numCols * sizeof(uchar4)); //make sure no memory is left laying around
	
	int threadSize = 16; //change to the right value
	int gridSizeX = (numCols + threadSize-1)/threadSize;
	int gridSizeY = (numRows + threadSize-1)/threadSize; 
	const dim3 blockSize(threadSize, threadSize, 1); 
	const dim3 gridSize(gridSizeX, gridSizeY, 1); 
	for (int i = 0; i < 30; i++){
		//row
		conv1DRow<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImageTemp, numRows, numCols);
		hipDeviceSynchronize();
		//col
      conv1DCol<<<gridSize, blockSize>>>(d_greyImageTemp, d_greyImage, numRows, numCols);
      hipDeviceSynchronize();
		//swap
		d_rgbaImage = d_greyImage;
	}

}

int main(int argc, char **argv) {
	hipDeviceReset();

	uchar4 *h_rgbaImage, *d_rgbaImage;
	uchar4 *h_greyImage, *d_greyImage;
	std::string input_file;
	std::string output_file;

	if (argc == 3) {
		input_file  = std::string(argv[1]);
		output_file = std::string(argv[2]);
	}
	else {
		std::cerr << "Usage: ./hw input_file output_file" << std::endl;
		exit(1);
	}

	//load the image and give us our input and output pointers
	preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, input_file);
	//call the students' code
	your_rgba_to_greyscale(h_rgbaImage, d_rgbaImage, d_greyImage, numRows(), numCols());
   hipDeviceSynchronize();
	hipGetLastError();
	printf("\n");
	postProcess(output_file); //prints gray image

   hipDeviceReset();
   return 0;
}
